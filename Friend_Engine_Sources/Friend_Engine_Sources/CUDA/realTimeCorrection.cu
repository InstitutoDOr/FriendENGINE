#include "hip/hip_runtime.h"
#include "RealTimeCorrection.cuh"
#include "legendre.h"

void printMatrix(int ncols, int n, const float*A, int lda, const char* name)
{
	for (int row = 0; row < ncols; row++){
		for (int col = 0; col < n; col++){
			double Areg = A[row + col*lda];
			//			printf("%s(%d,%d) = %f\n", name, row + 1, col + 1, Areg);
			printf("%5.5f\t", Areg);
		}
		printf("\n");
	}
}

void printMatrixT(int n, int ncols, const float*A, int lda, const char* name)
{
	for (int row = 0; row < n; row++){
		for (int col = 0; col < ncols; col++){
			double Areg = A[row*ncols + col];
			printf("%5.5f\t", Areg);
		}
		printf("\n");
	}
}

void fprintMatrix(int ncols, int n, const float*A, int lda, const char* name)
{
	FILE *f = fopen(name, "wt+");
	for (int row = 0; row < ncols; row++){
		for (int col = 0; col < n; col++){
			double Areg = A[row + col*lda];
			fprintf(f, "%5.5f,", Areg);
		}
		fprintf(f, "\n");
	}
	fclose(f);
}

void fprintMatrixT(int n, int ncols, const float*A, int lda, const char* name)
{
	FILE *f = fopen(name, "wt+");
	for (int row = 0; row < n; row++){
		for (int col = 0; col < ncols; col++){
			double Areg = A[row*ncols + col];
			fprintf(f, "%5.5f,", Areg);
		}
		fprintf(f, "\n");
	}
	fclose(f);
}

__global__ void computeTValue(int Ydim, int Ndata, float *Y, float *yret, float *contrastArray)
{
	int voxel = blockIdx.x * blockDim.x + threadIdx.x;
	if (voxel < Ydim)
	{
		float oldMean = 0, variance = 0, mean = 0;
		for (int volume = 0; volume < Ndata; volume++)
		{
			float value = Y[voxel*Ndata + volume];

			oldMean = mean;
			mean = mean + (value - oldMean) / (volume + 1);

			variance += (value - oldMean)*(value - mean);
		}
		variance /= (Ndata - 1);
		yret[voxel] = yret[voxel] / sqrt(variance * contrastArray[0]);
	}

}

__global__ void zeroConstantRow(int Ydim, int line, int xdim_cur, float *XtY)
{
	int voxel = blockIdx.x * blockDim.x + threadIdx.x;
	if (voxel < Ydim)
		XtY[line + voxel * xdim_cur] = 0;
}

/***************/
/* COPY KERNEL */
/***************/
__global__ void copy_kernel(const float * __restrict d_in1, float * __restrict d_out1, const float * __restrict d_in2, float * __restrict d_out2, const int M, const int N) {

	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < N) && (j < N)) {
		d_out1[j * N + i] = d_in1[j * M + i];
		d_out2[j * N + i] = d_in2[j * M + i];
	}
}

/* kernel function to normalize X */
__global__ void normalizeX(int m, int n, float *dev_X)
{
	/* Normalize devX to row mean=0, SD=100
	* Input:
	* 	m,n; row and column numbers of dev_X
	* 	dev_X; float pointer to device matrix
	* Output:
	* 	dev_X is overwritten by the normalized value
	*/

	float sum, ssum; // sum and squared sum
	float mean, sd; // mean and standard deviation
	int xi; // row index of X
	int j; // column index of X
	float x;

	xi = blockIdx.x * blockDim.x + threadIdx.x;
	if (xi >= m)
		return;

	sum = 0.0;
	ssum = 0.0;
	for (j = 0; j < n; j++) {
		x = *(dev_X + xi + (j * m));
		sum += x;
		ssum += x * x;
	}

	mean = sum / n;
	sd = sqrt(ssum / n - mean * mean);

	if (sd < FLT_EPSILON)
		return;

	for (j = 0; j < n; j++) {
		// subtract mean
		*(dev_X + xi + (j * m)) -= mean;
		// divide by sd x 100
		/* Calculation error could be large for small X values.
		* For keeping accuracy, X values are scaled to SD=100
		*/
		*(dev_X + xi + (j * m)) *= (100.0 / sd);
	}
};

/* kernel function to scaling Y */
__global__ void scalingY(int m, int n, float *dev_Ymeanscal, float *dev_Y) {
	/* Scaling devX to percent change
	* Input:
	* 	m,n; row and column numbers of dev_Y
	* 	dev_Ymeanscal; float pointer to device matrix of scaling factor
	* 	dev_Y; float pointer to device matrix of scaled value
	* Output:
	* 	dev_Y is overwritten by the scaled value
	*/

	int yi; // row index of Y
	int j; // column index of Y

	yi = blockIdx.x * blockDim.x + threadIdx.x;
	if (yi >= m)
		return;

	if (*(dev_Ymeanscal + yi) == 0.0) {
		float sum = 0.0;
		float mean;
		for (j = 0; j < n; j++)
			sum += *(dev_Y + yi + (j * m));

		mean = sum / float(n);
		if (mean >= FLT_EPSILON)
			*(dev_Ymeanscal + yi) = 100.0 / mean;
	}

	for (j = 0; j < n; j++)
		*(dev_Y + yi + (j * m)) *= *(dev_Ymeanscal + yi);
};


int RealTimeCorrection::rt_glm_cuda_init(int xdim, int ydim, int nmax, int rt_reg_ricor, int poly_order, float tr_sec)
{
	/* Initialize variables and keep memory
	* Input:
	* 	xdim, ydim; data dimensionality of X (regressor) and Y (regressed data)
	* 	nmax; maximum number of samples for GLM
	* 	rt_reg_ricor; bool flag for regress RVT and ricor
	* 	poly_order; maximum order of polynomial regressor
	* 	            (This may be overwritten by the estimated value with nmax)
	* 	tr_sec; Duration of one volume (sec)
	* 	NormX; bool flag for normalizing X (mean=0, std=1)
	* 	ScaleY; bool flag for scaling Y to percent change
	*/

	hipError_t cudaStat;
	/*--- Set parameters on global variables ---*/
	Ndata = 0;
	po_current = 0;
	Ydim = ydim;
	Nmax = nmax;
	TR_sec = tr_sec;
	ricor = rt_reg_ricor;
	data_xd = xdim;

	// Set regressor dimension
	Xdim = data_xd;

	// Add number of physiological noise regressors
	if (ricor) {
		row_ricor = Xdim;
		Xdim += 13; // Add RVT*5 + Resp*4 + ECG*4
	}

	// Update max_poly_order
	float len_sec = TR_sec * Nmax;
	int po = 1 + (int)(len_sec / 150.0); // polynomial order
	if (poly_order > po)
		max_poly_order = po;
	else
		max_poly_order = poly_order;

	// Add number of polynomial regressors to Xdim
	row_poly = Xdim;
	if (max_poly_order > 0)
		Xdim += max_poly_order + 1; // +1 is the baseline regressor (poly_order = 0)

	/*--- Allocate memory ---*/
	//Initialize pointers for conditional free
	X_buff = NULL;
	X = NULL;
	Y_buff = NULL;
	Y = NULL;
	Ymeanscal = NULL;
	XtX = NULL;
	iXtX = NULL;
	XtXa = NULL;
	iXtXa = NULL;
	XtY = NULL;
	info = NULL;
	contrast = NULL;

	cudaStat = hipMalloc((void**)&info, sizeof(int));
	CUDA_CHECK_RETURN(cudaStat);

	// XtX
	cudaStat = hipMalloc((void**)&XtX, Xdim * Xdim * sizeof(float));
	CUDA_CHECK_RETURN(cudaStat);

	cudaStat = hipMalloc((void**)&iXtX, Xdim * Xdim * sizeof(float));
	CUDA_CHECK_RETURN(cudaStat);

	// XtY
	cudaStat = hipMalloc((void**)&XtY, Xdim * Ydim * sizeof(float));
	CUDA_CHECK_RETURN(cudaStat);

	// XtX array
	cudaStat = hipMalloc((void**)&XtXa, sizeof(float *));
	CUDA_CHECK_RETURN(cudaStat);

	// iXtX array
	cudaStat = hipMalloc((void**)&iXtXa, sizeof(float *));
	CUDA_CHECK_RETURN(cudaStat);

	float **buff;
	buff = &XtX;
	hipMemcpy(XtXa, buff, sizeof(float *), hipMemcpyHostToDevice);

	buff = &iXtX;
	hipMemcpy(iXtXa, buff, sizeof(float *), hipMemcpyHostToDevice);

	// X_buff; ring buffer for X
	cudaStat = hipMalloc((void**)&X_buff, Xdim * Nmax * sizeof(float));
	CUDA_CHECK_RETURN(cudaStat);
	// Initialize with 0
	cudaStat = hipMemset((void*)X_buff, 0, Xdim * Nmax * sizeof(float));
	CUDA_CHECK_RETURN(cudaStat);

	// X
	cudaStat = hipMalloc((void**)&X, Xdim * Nmax * sizeof(float));
	CUDA_CHECK_RETURN(cudaStat);
	// Initialize with 0
	cudaStat = hipMemset((void*)X, 0, Xdim * Nmax * sizeof(float));
	CUDA_CHECK_RETURN(cudaStat);

	// Y_buff; ring buffer for Y
	cudaStat = hipMalloc((void**)&Y_buff, Ydim * Nmax * sizeof(float));
	CUDA_CHECK_RETURN(cudaStat);
	// Initialize with 0
	cudaStat = hipMemset((void*)Y_buff, 0, Ydim * Nmax * sizeof(float));
	CUDA_CHECK_RETURN(cudaStat);

	// Y
	cudaStat = hipMalloc((void**)&Y, Ydim * Nmax * sizeof(float));
	CUDA_CHECK_RETURN(cudaStat);
	// Initialize with 0
	cudaStat = hipMemset((void*)Y, 0, Ydim * Nmax * sizeof(float));
	CUDA_CHECK_RETURN(cudaStat);

	residuals = (float *) malloc(sizeof(float) * Nmax * Ydim);

	// Ymeanscal
	cudaStat = hipMalloc((void**)&Ymeanscal, Ydim * sizeof(float));
	CUDA_CHECK_RETURN(cudaStat);

	// Initialize with 0
	cudaStat = hipMemset((void*)Ymeanscal, 0, Ydim * sizeof(float));
	CUDA_CHECK_RETURN(cudaStat);

	// yret; for returning processed value vector of Y
	cudaStat = hipMalloc((void**)&yret, Ydim * sizeof(float));
	CUDA_CHECK_RETURN(cudaStat);

	if (max_poly_order > 0)
	{
		/*-- Set baseline value in X --*/
		float one = 1.0;
		int n;
		for (n = 0; n < Nmax; n++)
		{
			cudaStat = hipMemcpy(X_buff + row_poly + n * Xdim, &one, sizeof(float),
				hipMemcpyHostToDevice);
			CUDA_CHECK_RETURN(cudaStat);
		}
	}
	return 0;
}

int RealTimeCorrection::rt_glm_cuda_append_data(int t, float *x, float *y) {
	/* Add data vector x, y to X, Y
	* X and Y are saved in row-order for convenience of online update
	* Data size N is updated to t+1
	* Input:
	* 	t: point to insert data (0 base);
	*  x: pointer to vector of design matrix (Xdim length)
	*  y: pointer to vector of signal matrix (Ydim length)
	*/

	hipError_t cudaStat;

	Ndata = t + 1; //update the number of data

	/*--- Set data write position in the double buffer ---*/
	int buf_write_pos;
	buf_write_pos = t % Nmax;

	// Copy x at the buf_write_pos of dev_X_buff
	cudaStat = hipMemcpy(X_buff + buf_write_pos * Xdim, x,
		data_xd * sizeof(float), hipMemcpyHostToDevice);
	CUDA_CHECK_RETURN(cudaStat);

	// Copy y at the buf_end of dev_Y_buff
	cudaStat = hipMemcpy(Y_buff + (buf_write_pos * Ydim), y,
		Ydim * sizeof(float), hipMemcpyHostToDevice);
	CUDA_CHECK_RETURN(cudaStat);

	return 0;
}

int RealTimeCorrection::setContrastNumber(int count)
{
	hipError_t cudaStat;
	contrastCount = count;
	cudaStat = hipMalloc((void**)&contrast,  contrastCount * Xdim * sizeof(float));
	CUDA_CHECK_RETURN(cudaStat);

	cudaStat = hipMemset((void*)contrast, 0, contrastCount * Xdim * sizeof(float));
	CUDA_CHECK_RETURN(cudaStat);
	return 0;
}

int RealTimeCorrection::addContrast(float *contrst, int index, int len)
{
	hipError_t cudaStat; 
	cudaStat = hipMemcpy(contrast + index * Xdim, contrst, len * sizeof(float), hipMemcpyHostToDevice);
	CUDA_CHECK_RETURN(cudaStat);
	return 0;
}

int RealTimeCorrection::rt_glm_cuda_update_ricor(float *X_ricor, int len) {
	/* Update ricor regressors with len length
	* Input:
	*     X_ricor; pointer to that ricor regressor data (13 row * len column)
	*     len; length of regressor
	*/

	// If len is shorter than Ndata (if Ndata<Nmax) or Nmax, return error
	int data_len;
	if (Ndata < Nmax)
		data_len = Ndata;
	else
		data_len = Nmax;

	if (len < data_len) {
		fprintf(stderr, "RT: rt_glm_cuda, length of ricor is too short.\n");
		return -1;
	}

	// Copy ricor regressors
	int n;
	hipError_t cudaStat;
	for (n = 0; n < data_len; n++) {
		cudaStat = hipMemcpy(X_buff + row_ricor + n * Xdim, X_ricor + n * 13,
			sizeof(float) * 13, hipMemcpyHostToDevice);
		CUDA_CHECK_RETURN(cudaStat);
	}

	return 0;
}

int RealTimeCorrection::rt_glm_cuda_update_polyreg()
{
	/* Update polynomial regressors up to the current saved data length (Ndata)
	*/

	// If Ndata is not enough even for linear trend, return
	if (Ndata < 1)
		return -1;

	// Set length of regressor
	int polyreg_len;
	if (Ndata < Nmax)
		polyreg_len = Ndata;
	else
		polyreg_len = Nmax;

	// Set polynomial order
	if (polyreg_len < Nmax) {
		float len_sec = TR_sec * polyreg_len;
		po_current = 1 + (int)floor(len_sec / 150.0); // polynomial order
		if (po_current > max_poly_order)
			po_current = max_poly_order;
	}
	else
		po_current = max_poly_order;

	// Denominator for mapping 0..N_polyreg-1 into -1..1
	double aa = 2.0 / (polyreg_len - 1);
	double xx;

	// Append polynomial regressors
	int po, n;
	float poly_x;
	hipError_t cudaStat;
	for (po = 1; po < max_poly_order + 1; po++) {
		for (n = 0; n < polyreg_len; n++) {
			if (po <= po_current) {
				xx = aa * n - 1.0;
				poly_x = (float)legendre(xx, po);
			}
			else
				poly_x = 0.0;

			cudaStat = hipMemcpy(X_buff + row_poly + po + n * Xdim, &poly_x,
				sizeof(float), hipMemcpyHostToDevice);
			CUDA_CHECK_RETURN(cudaStat);
		}
	}

	return 0;
}

int RealTimeCorrection::rt_glm_cuda_get_residual(int t, int normX, int scaleY, float *YRet)
{
	/* Get residual vector of Y at t after regressing out X from Y
	* Input:
	* 	t: point of Y to get result (0 base)
	* 	normX; bool flag for normalizing X
	* 	scaleY; bool flag for scaling Y to percent change
	* Output:
	* 	yret: pointer to return vector (sizeof(float)*Ydim memory must be
	* 	      allocated)
	*/

	hipError_t cudaStat;
	char filename[100];
	int nthreads = 512;

	/*-- Check validity of t --*/
	if (t >= Ndata || t < Ndata - Nmax) { // Error in t
		fprintf(stderr, "RT: Error (rt_glm_cuda_get_residual); ");
		fprintf(stderr, "Time point %d is not in buffer ", t);
		if (Ndata < Nmax)
			fprintf(stderr, "(0-%d)\n", Ndata - 1);
		else
			fprintf(stderr, "(%d-%d)\n", Ndata - Nmax, Ndata - 1);

		memset(YRet, 0, sizeof(float) * Ydim);
		return -1;
	}

	/*-- Set length of data for regression --*/
	int N_reg; // Length of data for regression
	if (Ndata < Nmax)
		N_reg = Ndata;
	else
		N_reg = Nmax;

	// Current Xdim (exclude unset polynomial)
	int xdim_cur = Xdim - max_poly_order + po_current;

	hipblasStatus_t cublas_status;

	cublas_status = hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, Ndata, Xdim, &one, X_buff, Xdim, &zero, X_buff, Ndata, X, Ndata);
	cublas_status = hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, Ndata, Ydim, &one, Y_buff, Ydim, &zero, Y_buff, Ndata, Y, Ndata);

	if (1) //(outputFiles)
	{
		float *A = (float*)malloc(sizeof(float) * Xdim * Ndata);
		hipMemcpy(A, X, sizeof(float) * Ndata * Xdim, hipMemcpyDeviceToHost);

		sprintf(filename, "E:\\debug\\A_%d.csv", Ndata);
		fprintMatrix(Ndata, xdim_cur, A, Ndata, filename);

		free(A);
	}

	if (outputFiles)
	{
		float *A = (float*)malloc(sizeof(float) * Xdim * Ndata);
		float *B = (float*)malloc(sizeof(float) * Ydim * Ndata);
		hipMemcpy(A, X, sizeof(float) * Ndata * Xdim, hipMemcpyDeviceToHost);
		hipMemcpy(B, Y, sizeof(float) * Ndata * Ydim, hipMemcpyDeviceToHost);

		sprintf(filename, "E:\\debug\\A_%d.csv", Ndata);
		fprintMatrix(Ndata, Xdim, A, Ndata, filename);

		sprintf(filename, "E:\\debug\\B_%d.csv", Ndata);
		fprintMatrix(Ndata, Ydim, B, Ndata, filename);
		free(A);
		free(B);
	}

	dim3 threads = dim3(CUDANTHREAD, 1);
	dim3 blocks;

	/*-- Normalize X --*/
	if (normX == 1) {
		blocks = dim3(Xdim / threads.x, 1);
		normalizeX << <blocks, threads >> > (Xdim, N_reg, X);
	}

	/*-- Scale Y --*/
	if (scaleY == 1) {
		blocks = dim3(Ydim / threads.x, 1);
		scalingY << <blocks, threads >> >(Ydim, N_reg, Ymeanscal, Y);
	}

	cublas_status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, xdim_cur, xdim_cur, Ndata, &one, X, Ndata, X, Ndata, &zero, XtX, xdim_cur);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	cublas_status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, xdim_cur, Ydim, Ndata, &one, X, Ndata, Y, Ndata, &zero, XtY, xdim_cur);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	if (outputFiles)
	{
		float *A = (float*)malloc(sizeof(float) * Xdim * Xdim);
		float *B = (float*)malloc(sizeof(float) * Ydim * Xdim);
		hipMemcpy(A, XtX, sizeof(float) * xdim_cur * xdim_cur, hipMemcpyDeviceToHost);
		hipMemcpy(B, XtY, sizeof(float) * xdim_cur * Ydim, hipMemcpyDeviceToHost);

		sprintf(filename, "E:\\debug\\XtX_%d.csv", Ndata);
		fprintMatrix(xdim_cur, xdim_cur, A, xdim_cur, filename);

		sprintf(filename, "E:\\debug\\XtY_%d.csv", Ndata);
		fprintMatrix(xdim_cur, Ydim, B, xdim_cur, filename);
		free(A);
		free(B);
	}

	cublas_status = cublasSmatinvBatched(handle, xdim_cur, (const float **)XtXa, xdim_cur, iXtXa, xdim_cur, info, 1);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	if (outputFiles)
	{
		float *A = (float*)malloc(sizeof(float) * Xdim * Xdim);
		hipMemcpy(A, iXtX, sizeof(float) * xdim_cur * xdim_cur, hipMemcpyDeviceToHost);

		sprintf(filename, "E:\\debug\\iXtX_%d.csv", Ndata);
		fprintMatrix(xdim_cur, xdim_cur, A, xdim_cur, filename);

		free(A);
	}

	cublas_status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, xdim_cur, Ydim, xdim_cur, &one, iXtX, xdim_cur, XtY, xdim_cur, &zero, XtY, xdim_cur);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	if (outputFiles)
	{
		float *XCalc = (float*)malloc(sizeof(float) * Xdim * Ydim);
		cudaStat = hipMemcpy(XCalc, XtY, Ydim * Xdim * sizeof(float), hipMemcpyDeviceToHost);
		sprintf(filename, "E:\\debug\\X_%d.csv", Ndata);
		fprintMatrix(xdim_cur, Ydim, XCalc, xdim_cur, filename);
		free(XCalc);
	}

	if (calculateStatistics)
	{
		// residuals in Y
		cublas_status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Ndata, Ydim, xdim_cur, &alpha, X, Ndata, XtY, xdim_cur, &beta, Y, Ndata);
		assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

		// cT * Betas
		cublas_status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, contrastCount, Ydim, xdim_cur, &one, contrast, xdim_cur, XtY, xdim_cur, &zero, yret, contrastCount);
		assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

		if (outputFiles)
		{
			float *A = (float*)malloc(sizeof(float) * contrastCount * Ydim);
			float *B = (float*)malloc(sizeof(float) * Ydim * Ndata);
			hipMemcpy(A, yret, sizeof(float) * contrastCount * Ydim, hipMemcpyDeviceToHost);
			hipMemcpy(B, Y, sizeof(float) * Ndata * Ydim, hipMemcpyDeviceToHost);

			sprintf(filename, "E:\\debug\\CTBetas_%d.csv", Ndata);
			fprintMatrix(contrastCount, Ydim, A, contrastCount, filename);

			sprintf(filename, "E:\\debug\\Residuals_%d.csv", Ndata);
			fprintMatrix(Ndata, Ydim, B, Ndata, filename);
			free(A);
			free(B);
		}

		// cT * iXtX
		cublas_status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, contrastCount, xdim_cur, xdim_cur, &one, contrast, xdim_cur, iXtX, xdim_cur, &zero, XtX, xdim_cur);
		assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

		// anterior * c
		cublas_status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, contrastCount, xdim_cur, xdim_cur, &one, XtX, xdim_cur, contrast, xdim_cur, &zero, XtX, contrastCount);
		assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

		computeTValue <<<(Ydim + nthreads-1) / nthreads, nthreads>>> (Ydim, Ndata, Y, yret, XtX);

		cudaStat = hipMemcpy(YRet, yret, Ydim * sizeof(float), hipMemcpyDeviceToHost);
		CUDA_CHECK_RETURN(cudaStat);
	}
	else
	{
		// yret = B(t) - A(t)*X
		hipMemcpy(yret, Y_buff + ((Ndata - 1) * Ydim), sizeof(float) * Ydim, hipMemcpyDeviceToDevice);

		zeroConstantRow << <(Ydim + nthreads - 1) / nthreads, nthreads >> > (Ydim, row_poly, xdim_cur, XtY);
		if (outputFiles)
		{
			float *XCalc = (float*)malloc(sizeof(float) * Xdim * Ydim);
			cudaStat = hipMemcpy(XCalc, XtY, Ydim * Xdim * sizeof(float), hipMemcpyDeviceToHost);
			sprintf(filename, "E:\\debug\\Xdp_%d.csv", Ndata);
			fprintMatrix(xdim_cur, Ydim, XCalc, xdim_cur, filename);
			free(XCalc);
		}

		cublas_status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, Ydim, xdim_cur, &alpha, X_buff + ((Ndata - 1) * Xdim), 1, XtY, xdim_cur, &beta, yret, 1);
		assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

		// Copy back dev_yret to host Ys
		cudaStat = hipMemcpy(YRet, yret, Ydim * sizeof(float), hipMemcpyDeviceToHost);
		CUDA_CHECK_RETURN(cudaStat);
		if (outputFiles)
		{
			sprintf(filename, "E:\\debug\\BRTC_%d.csv", Ndata);
			fprintMatrix(1, Ydim, YRet, 1, filename);
		}

	}
	return 0;
}

/*----------------------------------------------------------------------------*/
int RealTimeCorrection::rt_glm_cuda_free_GPU_memory()
{
	/* Free GPU memory */

	hipError_t cudaStat;

	if (X_buff) {
		cudaStat = hipFree(X_buff);
		CUDA_CHECK_RETURN(cudaStat);
	}

	if (Y_buff) {
		cudaStat = hipFree(Y_buff);
		CUDA_CHECK_RETURN(cudaStat);
	}

	if (X) {
		cudaStat = hipFree(X);
		CUDA_CHECK_RETURN(cudaStat);
	}

	if (Y) {
		cudaStat = hipFree(Y);
		CUDA_CHECK_RETURN(cudaStat);
	}

	if (Ymeanscal) {
		cudaStat = hipFree(Ymeanscal);
		CUDA_CHECK_RETURN(cudaStat);
	}

	if (yret) {
		cudaStat = hipFree(yret);
		CUDA_CHECK_RETURN(cudaStat);
	}

	if (XtX) hipFree(XtX);
	if (XtY) hipFree(XtY);

	if (iXtX) hipFree(iXtX);
	if (iXtXa) hipFree(iXtXa);
	if (XtXa) hipFree(XtXa);
	if (info) hipFree(info);
	if (contrast) hipFree(contrast);
	if (residuals) free(residuals);

	return 0;
}


/*----------------------------------------------------------------------------*/
int RealTimeCorrection::rt_glm_cuda_get_Ndata() {
	/* Return number of data points saved in the library */
	return Ndata;
}

/*----------------------------------------------------------------------------*/
/* Print matrix values on device (for debug) */
int RealTimeCorrection::_print_device_Mtx(int n, int m, float* dev_Mtx)
{
	float *host_Mtx = (float *)malloc(n * m * sizeof(float));

	hipError_t cudaStat;
	cudaStat = hipMemcpy(host_Mtx, dev_Mtx, n * m * sizeof(float),
		hipMemcpyDeviceToHost);
	CUDA_CHECK_RETURN(cudaStat);

	int i, j;
	for (i = 0; i < n; i++) {
		for (j = 0; j < m; j++)
			printf("%f ", host_Mtx[i + j * n]);

		printf("\n");
	}

	free(host_Mtx);
	return 0;
}

/*----------------------------------------------------------------------------*/
void RealTimeCorrection::print_X_on_GPU()
{
	int N_print;
	if (Ndata < Nmax * 2)
		N_print = Ndata;
	else
		N_print = Nmax * 2;

	_print_device_Mtx(Xdim, N_print, X);
}

void RealTimeCorrection::print_Y_on_GPU(void)
{
	int N_print;
	if (Ndata < Nmax * 2)
		N_print = Ndata;
	else
		N_print = Nmax * 2;

	_print_device_Mtx(Ydim, N_print, Y);
}

/*----------------------------------------------------------------------------*/
/* Get matrix values on device */
int RealTimeCorrection::_get_device_Mtx(int n, int m, float* dev_Mtx, float* host_Mtx)
{
	hipError_t cudaStat;
	cudaStat = hipMemcpy(host_Mtx, dev_Mtx, n * m * sizeof(float),
		hipMemcpyDeviceToHost);
	CUDA_CHECK_RETURN(cudaStat);

	return 0;
}

/*----------------------------------------------------------------------------*/
int RealTimeCorrection::get_Xdim_on_GPU()
{
	return Xdim;
}

int RealTimeCorrection::get_Ydim_on_GPU()
{
	return Ydim;
}

void RealTimeCorrection::get_X_on_GPU(float* host_Mtx)
{
	_get_device_Mtx(Xdim, Nmax * 2, X, host_Mtx);
}

void RealTimeCorrection::get_Y_on_GPU(float* host_Mtx)
{
	_get_device_Mtx(Ydim, Nmax * 2, Y, host_Mtx);
}

void RealTimeCorrection::get_beta_on_GPU(float* host_Mtx)
{
	//	_get_device_Mtx(Xdim, Ydim, XtY, host_Mtx);
}

int RealTimeCorrection::init_cuda()
{
	status = hipblasCreate(&handle);

	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "!!!! CUBLAS initialization error\n");
		return EXIT_FAILURE;
	}
	return 0;
}

int RealTimeCorrection::end_cuda()
{
	/* Shutdown */
	status = hipblasDestroy(handle);

	// hipDeviceReset causes the driver to clean up all state. While
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling hipDeviceReset causes all profile data to be
	// flushed before the application exits
	hipDeviceReset();
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "!!!! shutdown error (A)\n");
		return EXIT_FAILURE;
	}
	return 0;
}

int RealTimeCorrection::canGetResidual()
{
	//int xdim_cur = Xdim - max_poly_order + po_current;
	//if (Ndata < Xdim) return 0;
	if (Ndata < 50) return 0;
	return 1;
}